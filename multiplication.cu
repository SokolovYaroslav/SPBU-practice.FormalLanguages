#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <iostream>
#include <string>
#include "multiplication.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
		{
			fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define largest_pow2 32
#define threads_x 32

__device__ bool is_changed;

size_t matrix_memsize;
uint32_t *tmp_matrix;
hipEvent_t start;
hipEvent_t stop;

void initialize(int N_inp) {
	int dev = findCudaDevice(argc, (const char **)argv);

	N = N_inp;
	rows = N;
	cols = N / largest_pow2 + (N % largest_pow2 ? 1 : 0);
	matrix_memsize = rows * cols * sizeof(uint32_t);

	gpuErrchk(hipMalloc(reinterpret_cast<void **>(&tmp_matrix), matrix_memsize));
	gpuErrchk(hipEventCreate(&start));
	gpuErrchk(hipEventCreate(&stop));
}

__global__ void DummyMulAdd(uint32_t *A, uint32_t *B, uint32_t *C, int cols) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y;

    if (x >= cols) {
        return;
    }

	uint32_t acc = 0;
	uint32_t a_el;
	for (uint32_t i = 0; i < cols; ++i) {
		a_el = A[y * cols + i];
		#pragma unroll
		for (uint32_t b = 0; b < 32; ++b) {
			if (a_el & 1) {
				acc |= B[x + 32 * cols * i + cols * (31 - b)];
			}
			a_el >>= 1;
		}
	}

	uint32_t c_old = C[y * cols + x];
	if (c_old != (acc | c_old)) {
		is_changed = true;
		C[y * cols + x] = acc | c_old;
	}
}

__global__ void DummyMul(uint32_t *A, uint32_t *B, uint32_t *C, int cols) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y;

    if (x >= cols) {
        return;
    }

	uint32_t acc = 0;
	uint32_t a_el;
	for (uint32_t i = 0; i < cols; ++i) {
		a_el = A[y * cols + i];
		#pragma unroll
		for (uint32_t b = 0; b < 32; ++b) {
			if (a_el & 1) {
				acc |= B[x + 32 * cols * i + cols * (31 - b)];
			}
			a_el >>= 1;
		}
	}

	C[y * cols + x] = acc;
}


__global__ void AddToLeft(uint32_t *A, uint32_t *B, int cols) {
	int index = blockIdx.y * cols + blockIdx.x * blockDim.x + threadIdx.x;

    if (blockIdx.x * blockDim.x + threadIdx.x >= cols) {
        return;
    }

	uint32_t A_el = A[index];
	uint32_t res = B[index] | A_el;
	if (res != A_el) {
		is_changed = true;
		A[index] = res;
	}
}

void wait_() {
	hipDeviceSynchronize();
}

gpuErrchk(hipEventRecord(stop, NULL));

// Wait for the stop event to complete
gpuErrchk(hipEventSynchronize(stop));

float msecTotal = 0.0f;
gpuErrchk(hipEventElapsedTime(&msecTotal, start, stop));

// Compute and print the performance
float msecPerMatrixMul = msecTotal / nIter;
printf("Time= %.3f msec\n", msecPerMatrixMul);


void start_time() {
	gpuErrchk(hipEventRecord(start, NULL));
}

void stop_time() {
	gpuErrchk(hipEventRecord(stop, NULL));
	gpuErrchk(hipEventSynchronize(stop));

	float msecTotal = 0.0f;
	gpuErrchk(hipEventElapsedTime(&msecTotal, start, stop));

	float msecPerMatrixMul = msecTotal / nIter;
	printf("Time= %.3f msec\n", msecPerMatrixMul);
}

void gpuMemset(uint32_t *d_M, int val) {
	gpuErrchk(hipMemset(d_M, val, matrix_memsize));
}

uint32_t * device_matrix_alloc() {
	uint32_t *d_M;
	gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_M), matrix_memsize));

	return d_M;
}

uint32_t * host_matrix_calloc() {
    uint32_t *M;
    gpuErrchk(hipHostMalloc(reinterpret_cast<void **>(&M), matrix_memsize));
    gpuMemset(M, 0);
	return M;
}

void gpu2cpu(uint32_t *d_M, uint32_t *h_M) {
	gpuErrchk(hipMemcpyAsync(h_M, d_M, matrix_memsize, hipMemcpyDeviceToHost));
}

void cpu2gpu(uint32_t *h_M, uint32_t *d_M) {
	gpuErrchk(hipMemcpyAsync(d_M, h_M, matrix_memsize, hipMemcpyHostToDevice));
}

void setFlag() {
	bool flag = false;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(is_changed), &flag, sizeof(bool)))
}

uint8_t getFlag() {
	bool flag;
	gpuErrchk(hipMemcpyFromSymbol(&flag, HIP_SYMBOL(is_changed), sizeof(bool)))

	return flag;
}

bool MatrixMulAdd(uint32_t *A, uint32_t *B, uint32_t *C) {
	bool safe = (A == C) || (B == C);
	dim3 mul_threads(threads_x);
	dim3 mul_blocks(cols / threads_x + (cols % threads_x ? 1 : 0), rows);

    setFlag();
	if (safe) {
		DummyMul <<<mul_blocks, mul_threads>>> (A, B, tmp_matrix, cols);
		hipDeviceSynchronize();
		gpuErrchk(hipGetLastError());
		AddToLeft <<<mul_blocks, mul_threads>>> (C, tmp_matrix, cols);
		hipDeviceSynchronize();
		gpuErrchk(hipGetLastError());
	}
	else {
		DummyMulAdd <<<mul_blocks, mul_threads>>> (A, B, C, cols);
		hipDeviceSynchronize();
		gpuErrchk(hipGetLastError());
	}

	return getFlag();
}
