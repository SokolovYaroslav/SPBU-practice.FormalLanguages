#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <iostream>
#include <string>
#include <ctime>
#include "multiplication.h"

#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
		{
			fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define BITS_PER_TYPE 32
#define THREADS_PER_BLOCK 32 // More is faster if N > 5000

__device__ bool is_changed;

size_t matrix_memsize;
uint32_t *tmp_matrix;
clock_t begin;
clock_t end;

void initialize(int N_inp) {
	int devID = 0;
	devID = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(devID));

	N = N_inp;
	rows = N;
	cols = N / BITS_PER_TYPE + (N % BITS_PER_TYPE ? 1 : 0);
	matrix_memsize = rows * cols * sizeof(uint32_t);

	gpuErrchk(hipMalloc(reinterpret_cast<void **>(&tmp_matrix), matrix_memsize));
}

__global__ void DummyMulAdd(uint32_t *A, uint32_t *B, uint32_t *C, int cols) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y;

    if (x >= cols) {
        return;
    }

	uint32_t acc = 0;
	uint32_t a_el;
	for (uint32_t i = 0; i < cols; ++i) {
		a_el = A[y * cols + i];
		#pragma unroll
		for (uint32_t b = 0; b < 32; ++b) {
			if (a_el & 1) {
				acc |= B[x + 32 * cols * i + cols * (31 - b)];
			}
			a_el >>= 1;
		}
	}

	if (acc == 0) {
		return;
	}

	uint32_t c_old = C[y * cols + x];
	if (c_old != (acc | c_old)) {
		is_changed = true;
		C[y * cols + x] = acc | c_old;
	}
}

__global__ void DummyMul(uint32_t *A, uint32_t *B, uint32_t *C, int cols) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y;

    if (x >= cols) {
        return;
    }

	uint32_t acc = 0;
	uint32_t a_el;
	for (uint32_t i = 0; i < cols; ++i) {
		a_el = A[y * cols + i];
		#pragma unroll
		for (uint32_t b = 0; b < 32; ++b) {
			if (a_el & 1) {
				acc |= B[x + 32 * cols * i + cols * (31 - b)];
			}
			a_el >>= 1;
		}
	}

	C[y * cols + x] = acc;
}


__global__ void AddToLeft(uint32_t *A, uint32_t *B, int cols) {
	int index = blockIdx.y * cols + blockIdx.x * blockDim.x + threadIdx.x;

    if ((blockIdx.x * blockDim.x + threadIdx.x) >= cols) {
        return;
    }

	uint32_t A_el = A[index];
	uint32_t res = B[index] | A_el;
	if (res != A_el) {
		is_changed = true;
		A[index] = res;
	}
}

void wait_() {
	hipDeviceSynchronize();
}

void start_time() {
	begin = clock();
}

void stop_time() {
	end = clock();
	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	printf("%d\n", (int) (elapsed_secs * 1000 + 0.5));
}

void gpuMemset(uint32_t *d_M, int val) {
	gpuErrchk(hipMemset(d_M, val, matrix_memsize));
}

uint32_t * device_matrix_alloc() {
	uint32_t *d_M;
	gpuErrchk(hipMalloc(reinterpret_cast<void **>(&d_M), matrix_memsize));

	return d_M;
}

uint32_t * host_matrix_calloc() {
    uint32_t *M;
    gpuErrchk(hipHostMalloc(reinterpret_cast<void **>(&M), matrix_memsize));
    gpuMemset(M, 0);
	return M;
}

void gpu2cpu_async(uint32_t *d_M, uint32_t *h_M) {
	gpuErrchk(hipMemcpyAsync(h_M, d_M, matrix_memsize, hipMemcpyDeviceToHost));
}

void cpu2gpu_async(uint32_t *h_M, uint32_t *d_M) {
	gpuErrchk(hipMemcpyAsync(d_M, h_M, matrix_memsize, hipMemcpyHostToDevice));
}

void setFlag() {
	bool flag = false;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(is_changed), &flag, sizeof(bool)))
}

uint8_t getFlag() {
	bool flag;
	gpuErrchk(hipMemcpyFromSymbol(&flag, HIP_SYMBOL(is_changed), sizeof(bool)))

	return flag;
}

bool MatrixMulAdd(uint32_t *A, uint32_t *B, uint32_t *C) {
	bool safe = (A == C) || (B == C);
	dim3 mul_threads(THREADS_PER_BLOCK);
	dim3 mul_blocks(cols / THREADS_PER_BLOCK + (cols % THREADS_PER_BLOCK ? 1 : 0), rows);

    setFlag();
	if (safe) {
		DummyMul <<<mul_blocks, mul_threads>>> (A, B, tmp_matrix, cols);
		hipDeviceSynchronize();
		gpuErrchk(hipGetLastError());
		AddToLeft <<<mul_blocks, mul_threads>>> (C, tmp_matrix, cols);
		hipDeviceSynchronize();
		gpuErrchk(hipGetLastError());
	}
	else {
		DummyMulAdd <<<mul_blocks, mul_threads>>> (A, B, C, cols);
		hipDeviceSynchronize();
		gpuErrchk(hipGetLastError());
	}

	return getFlag();
}
